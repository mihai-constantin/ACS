#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

/*
 *	hash function
 */
__device__ int getHashCode(int key, int capacity) {
	return ((long long)abs(key) * 1646237llu) % 67965551447llu % capacity;
}

__global__ void kernel_insert(int *keys, int *values, int N, HashTable hashtable) {

	/* compute the global element index this thread should process */
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	/* avoid accessing out of bounds elements */
	if (i < N) {

		/* get start index for current key based by current capacity of the hashtable */
		int idx = getHashCode(keys[i], hashtable.capacity);

		/* 
		 * iterate through hashtable to find a free slot for insertion 
		 * use atomicCAS for threads synchronization
		 * if current key is 0, set its key to keys[i] and its value to values[i]
		 * if current key is keys[i], update its value
		 */

		/* iterate through [idx, hashtable.capacity) positions */
		for (int k = idx; k < hashtable.capacity; k++) {
			int old_key = atomicCAS(&hashtable.nodes[k].key, 0, keys[i]);
			if (!old_key || old_key == keys[i]) {
				hashtable.nodes[k].value = values[i];
				return;
			}
		}

		/* iterate through [0, idx) positions */
		for (int k = 0; k < idx; k++) {
			int old_key = atomicCAS(&hashtable.nodes[k].key, 0, keys[i]);
			if (!old_key || old_key == keys[i]) {
				hashtable.nodes[k].value = values[i];
				return;
			}
		}
	}
}

__global__ void kernel_reshape(HashTable old_hashtable, HashTable new_hashtable) {

	/* compute the global element index this thread should process */
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	/* avoid accessing out of bounds elements */
	if (i < old_hashtable.capacity) {

		/* verify if there is a key at index */
		if (old_hashtable.nodes[i].key != 0) {

			/* get start index for current key based by capacity of the new hashtable */
			int idx = getHashCode(old_hashtable.nodes[i].key, new_hashtable.capacity);

			/* 
			 * iterate through hashtable to find a free slot for insertion 
			 * use atomicCAS for threads synchronization
			 */
			 
			/*
			 * if current key from new_hashtable is 0, 
			 * set its key to old_hashtable.nodes[i].key 
			 * and its value to old_hashtable.nodes[i].value
			 */

			/* iterate through [idx, new_hashtable.capacity) positions */
			for (int k = idx; k < new_hashtable.capacity; k++) {
				int old_key = atomicCAS(&new_hashtable.nodes[k].key, 0, old_hashtable.nodes[i].key);
	
				if (!old_key) {
					new_hashtable.nodes[k].value = old_hashtable.nodes[i].value;
					return;
				}
			}
	
			/* iterate through [0, idx) positions */
			for (int k = 0; k < idx; k++) {
				int old_key = atomicCAS(&new_hashtable.nodes[k].key, 0, old_hashtable.nodes[i].key);
	
				if (!old_key) {
					new_hashtable.nodes[k].value = old_hashtable.nodes[i].value;
					return;
				}
			}
		}
	}
}

__global__ void kernel_get(int *keys, int *values, int N, HashTable hashtable) {
	/* compute the global element index this thread should process */
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	/* avoid accessing out of bounds elements */
	if (i < N) {

		/* get start index for current key based by current capacity of the hashtable */
		int idx = getHashCode(keys[i], hashtable.capacity);

		/* 
		 * iterate through hashtable to find the key
		 * set values[i] to corresponding value of the key
		 */

		/* iterate through [idx, hashtable.capacity) positions */
		for (int k = idx; k < hashtable.capacity; k++) {
			if (hashtable.nodes[k].key == keys[i]) {
				values[i] = hashtable.nodes[k].value;
				return;
			}
		}

		/* iterate through [0, idx) positions */
		for (int k = 0; k < idx; k++) {
			if (hashtable.nodes[k].key == keys[i]) {
				values[i] = hashtable.nodes[k].value;
				return;
			}
		}
	}

}

/* INIT HASH */
GpuHashTable::GpuHashTable(int size) {

	/* initially, we don't have any elements inserted into hashtable */
	hashtable.size = 0;
	
	/* set initial capacity to given size parameter */
	hashtable.capacity = size;

	const int num_bytes = size * sizeof(Node);

	/* alloc memory for nodes vector */
	hipMalloc(&hashtable.nodes, num_bytes);
	if (!hashtable.nodes) {
		return;
	}

	/* initially, set nodes vector to 0 */
	hipMemset(hashtable.nodes, 0, num_bytes);
}

/* DESTROY HASH */
GpuHashTable::~GpuHashTable() {
	/* free memory used by nodes vector */
	hipFree(hashtable.nodes);
}

/* RESHAPE HASH */
void GpuHashTable::reshape(int numBucketsReshape) {

	HashTable new_hashtable;
	
	/* set new capacity such that load factor remains to 80% */	
	new_hashtable.capacity = (int)(numBucketsReshape / 0.8f);

	/* set new size to current hashtable's size */
	new_hashtable.size = hashtable.size;

	const int num_bytes = new_hashtable.capacity * sizeof(Node);

	/* alloc memory for nodes vector */
	hipMalloc(&new_hashtable.nodes, num_bytes);
	if (!new_hashtable.nodes) {
		return;
	}

	/* initially, set nodes vector to 0 */
	hipMemset(new_hashtable.nodes, 0, num_bytes);

	/*
	 *	Compute the parameters necessary to run the kernel
	 *	- the number of blocks
	 *  - the number of threads per block 
	 */

	unsigned int blocks_no = hashtable.capacity / 1024;

	/* deal with a possible partial final block */
	if (hashtable.capacity % 1024) {
		++blocks_no;
	}

	/* execute the kernel */
	kernel_reshape <<< blocks_no, 1024 >>> (hashtable, new_hashtable);

	hipDeviceSynchronize();

	/* free memory used by nodes vector */
	hipFree(hashtable.nodes);

	/* set hashtable to new_hashtable */
	hashtable = new_hashtable;
}

/* INSERT BATCH */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {

	const int num_bytes = numKeys * sizeof(int);

	int *device_keys = 0;
	int *device_values = 0;

	/* allocate the device's arrays */
	hipMalloc(&device_keys, num_bytes);
	hipMalloc(&device_values, num_bytes);

	if (!device_keys || !device_values) {
		/* could not allocate enough memory */
		return false;
	}

	/* reshape if necessary */
	float new_factor = (float)(hashtable.size + numKeys) / (float)hashtable.capacity;
	if (new_factor > 0.8f) {
		reshape(hashtable.size + numKeys);
	}

	/* copy from host(cpu/ram) to device(gpu/vram) */
	hipMemcpy(device_keys, keys, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, num_bytes, hipMemcpyHostToDevice);

	/*
	 *	Compute the parameters necessary to run the kernel
	 *	- the number of blocks
	 *  - the number of threads per block 
	 */
	
	unsigned int blocks_no = numKeys / 1024;

	/* deal with a possible partial final block */
	if (numKeys % 1024) {
		++blocks_no;
	}

	/* execute the kernel */
	kernel_insert<<< blocks_no, 1024 >>> (device_keys, 
											device_values, 
											numKeys, 
											hashtable);

	hipDeviceSynchronize();

	/* increase the size of hashtable with numKeys */
	hashtable.size += numKeys;

	/* free memory */
	hipFree(device_keys);
	hipFree(device_values);

	return true;
}

/* GET BATCH */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	int *host_values = 0;
	int *device_keys = 0;
	int *device_values = 0;

	const int num_bytes = numKeys * sizeof(int);

	/* allocate the host_values array */
	host_values = (int *) malloc(num_bytes);

	/* allocate the device's arrays */
	hipMalloc(&device_keys, num_bytes);
	hipMalloc(&device_values, num_bytes);

	if(!host_values || !device_keys || !device_values) {
		/* could not allocate enough memory */
		return NULL;
	}

	/* copy from host(cpu/ram) to device(gpu/vram) */
	hipMemcpy(device_keys, keys, num_bytes, hipMemcpyHostToDevice);

	/*
	 *	Compute the parameters necessary to run the kernel
	 *	- the number of blocks
	 *  - the number of threads per block 
	 */

	unsigned int blocks_no = numKeys / 1024;
	
	/* deal with a possible partial final block */
	if (numKeys % 1024) {
		++blocks_no;
	}

	/* execute the kernel */
	kernel_get<<< blocks_no, 1024 >>> (device_keys,
										device_values, 
										numKeys, 
										hashtable);

	hipDeviceSynchronize();

	/* copy from device(gpu/vram) to host(cpu/ram) */
	hipMemcpy(host_values, device_values, num_bytes, hipMemcpyDeviceToHost);

	return host_values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	return (float)hashtable.size / (float)hashtable.capacity;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
