

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

#define TILE_WIDTH 16

// Task 1 - simple matrix multiplication
__global__ void matrix_multiply_simple(float *ma, float *mb, float *mc, size_t width)
{
	// calculate the row & column index of the element
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row * width + col;

	// do dot product between row of ma and column of mb
	float res = 0;
	for (int i = 0; i < width; i++) {
		res += ma[row * width + i] * mb[i * width + col];
	}

	// write result in mc
	mc[idx] = res;
}

// Task 2 - optimized matrix multiplication
__global__ void matrix_multiply(float *ma, float *mb, float *mc, size_t width)
{
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x,  by = blockIdx.y;

	// allocate 2D tiles in __shared__ memory
	__shared__ float ma_tile[TILE_WIDTH][TILE_WIDTH];
	__shared__ float mb_tile[TILE_WIDTH][TILE_WIDTH];

	// calculate the row & column index of the element
	int row = ty + by * blockDim.y;
	int col = tx + bx * blockDim.x;
	int idx = row * width + col;

	float result = 0;

	// loop over the tiles of the input
	for(int t = 0; t < width/TILE_WIDTH; ++t) {
	
		// load tiles into __shared__ memory allocated before
		ma_tile[ty][tx] = ma[row * width + t * TILE_WIDTH + tx];
		mb_tile[ty][tx] = mb[(t * TILE_WIDTH + ty) * width + col];
		
		// wait until all data is loaded before allowing
		// any thread in this block to continue
		__syncthreads();

		// do dot product between row of tile from ma and column of tile from mb
		for (int i = 0; i < TILE_WIDTH; i++) {
			result += ma_tile[ty][i] * mb_tile[i][tx];
		}

		// wait until all data is loaded before allowing
		// any thread in this block to continue
		__syncthreads();
	}

	// write result in mc
	mc[idx] = result;
}

int main(void)
{
	// create a large workload so we can easily measure the
	// performance difference of both implementations

	// note that n measures the width of the matrix, not the number of total elements
	const size_t n = 1 << 10;
	const dim3 block_size(TILE_WIDTH,TILE_WIDTH);
	const dim3 num_blocks(n / block_size.x, n / block_size.y);

	// generate random input on the host
	std::vector<float> host_a(n*n), host_b(n*n), host_c(n*n);
	for(int i = 0; i < n * n; ++i) {
		host_a[i] = static_cast<float>(rand()) / RAND_MAX;
		host_b[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	// allocate storage for the device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void**)&device_a, sizeof(float) * n * n);
	hipMalloc((void**)&device_b, sizeof(float) * n * n);
	hipMalloc((void**)&device_c, sizeof(float) * n * n);

	// copy input to the device
	hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n, hipMemcpyHostToDevice);

	//Task 3 - measure the time spent in the kernel for simple and optimized implementation
	
	// create CUDA events for measuring kernel time
	hipEvent_t launch_begin, launch_end;
	hipEventCreate(&launch_begin);
	hipEventCreate(&launch_end);

	// time many kernel launches and take the average time
	const size_t num_launches = 100;
	float average_simple_time = 0;
	std::cout << "Timing simple implementation...";
	
	for(int i = 0; i < num_launches; ++i) {
		// record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin, 0);
		matrix_multiply_simple<<<num_blocks, block_size>>>(device_a, device_b, device_c, n);
		hipEventRecord(launch_end, 0);

		// Wait for launch_end event to complete
		hipEventSynchronize(launch_end);

		// measure the time spent in the kernel
		float time = 0;
		hipEventElapsedTime(&time, launch_begin, launch_end);

		average_simple_time += time;
	}
	
	average_simple_time /= num_launches;
	std::cout << " done." << std::endl;

	//now time the optimized kernel

	// time many kernel launches and take the average time
	float average_optimized_time = 0;
	std::cout << "Timing optimized implementation...";
	for(int i = 0; i < num_launches; ++i) {
		// record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin, 0);
		matrix_multiply<<<num_blocks, block_size>>>(device_a, device_b, device_c, n);
		hipEventRecord(launch_end, 0);

		// Wait for launch_end event to complete
		hipEventSynchronize(launch_end);
		
		// measure the time spent in the kernel
		float time = 0;
		hipEventElapsedTime(&time, launch_begin, launch_end);

		average_optimized_time += time;
	}
	average_optimized_time /= num_launches;
	std::cout << " done." << std::endl;

	// report the effective throughput of each kernel in GFLOPS
	// the effective throughput is measured as the number of floating point operations performed per second:
	// (one mul + one add) * N^3
	float simple_throughput = static_cast<float>(2 * n * n * n) / (average_simple_time / 1000.0f) / 1000000000.0f;
	float optimized_throughput = static_cast<float>(2 * n * n * n) / (average_optimized_time / 1000.0f) / 1000000000.0f;

	std::cout << "Matrix size: " << n << "x" << n << std::endl;
	std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

	std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GFLOPS" << std::endl;
	std::cout << "Performance improvement: " << optimized_throughput / simple_throughput << "x" << std::endl;
	std::cout << std::endl;

	// destroy the CUDA events
	hipEventDestroy(launch_begin);
	hipEventDestroy(launch_end);

	// deallocate device memory
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return 0;
}

