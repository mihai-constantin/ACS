
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

static void HandleError(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " 
            << file << " at line " << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//#define MAGNITUDE       (1)
#define MAGNITUDE       (1024 * 1024)
#define NUM_BLOCKS      8 * MAGNITUDE
#define NUM_THREADS     16
#define NUM_ELEM        100 * MAGNITUDE

__global__ void kernel_compute(int* data) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // invalid access
    data[idx] = 1111 * idx;
}

int main(int argc, char *argv[]) {
    int* data = NULL;

    HANDLE_ERROR( hipMalloc(&data, 1 * sizeof(int)) );

    // launch kernel
    kernel_compute<<<NUM_BLOCKS, NUM_THREADS>>>(data);
    HANDLE_ERROR( hipDeviceSynchronize() );

    return 0;
}