#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    
    // global element index for current thread
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(i < N) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    const int num_elements = N;
    const int num_bytes = num_elements * sizeof(float);

    // TODO 1: Allocate the host's arrays
    host_array_a = (float *) malloc(num_bytes);
    host_array_b = (float *) malloc(num_bytes);
    host_array_c = (float *) malloc(num_bytes);

    // TODO 2: Allocate the device's arrays
    hipMalloc((void **) &device_array_a, num_bytes);
    hipMalloc((void **) &device_array_b, num_bytes);
    hipMalloc((void **) &device_array_c, num_bytes);

    // TODO 3: Check for allocation errors
    if(!host_array_a || !host_array_b || !host_array_c || 
        !device_array_a || !device_array_b || !device_array_c) {
        printf("[HOST] Couldn't allocate memory\n");
        return 1;
    }

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.

    fill_array_float(host_array_a, num_elements);
    fill_array_random(host_array_b, num_elements);

    // TODO 5: Copy the host's arrays to device
    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    const size_t block_size = 256;
    size_t blocks_no = num_elements / block_size;

    if (num_elements % block_size)
        ++blocks_no;

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).

    add_arrays<<<blocks_no, block_size>>>(device_array_a, device_array_b, device_array_c, num_elements);
    hipDeviceSynchronize();

    // TODO 7: Copy back the results and then uncomment the checking function
    hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
    free(host_array_a);
    free(host_array_b);
    free(host_array_c);
    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);
   
    return 0;
}