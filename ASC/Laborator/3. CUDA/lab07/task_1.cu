#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {

    // global element index for current thread
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i < N) {

        a[i] = a[i] % 2;
    }

}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {

    // global element index for current thread
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (i < N) {
        a[i] = blockIdx.x;
    }
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {

    // global element index for current thread
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i < N) {
        a[i] = threadIdx.x;
    }
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);
    printf("[HOST] You have %d CUDA-capable GPU(s)\n", nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("Device number: %d\n", i);
        printf("\tDevice name: %s\n", deviceProp.name);
        printf("\tTotal global memory: %zu\n", deviceProp.totalGlobalMem);
        printf("\tMemory Clock Rate(KHz): %d\n", deviceProp.clockRate);
        printf("\tMemory Bus Width (bits): %d\n", deviceProp.memoryBusWidth);
    }

    hipSetDevice(0);

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.
    const int num_elements = 16;
    const int num_bytes = num_elements * sizeof(int);

    // declaring host and device arrays
    int *host_array = 0;
    int *device_array = 0;

    // alocate the host array
    host_array = (int *) malloc(num_bytes);

    // alocate the device's array
    hipMalloc((void **) &device_array, num_bytes);

    if (!host_array || !device_array) {
        printf("[HOST] Couldn't allocate memory\n");
        return 1;
    }

    // initialize the host array
    fill_array_int(host_array, num_elements);

    // copy from host(cpu/ram) to device(gpu/vram)
    hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //

    kernel_parity_id<<<4, 4>>>(device_array, num_elements);
    hipDeviceSynchronize();
    
    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(3, host_array);

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //

    kernel_block_id<<<4,4>>>(device_array, num_elements);
    hipDeviceSynchronize();

    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(4, host_array);

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //

    kernel_thread_id<<<4, 4>>>(device_array, num_elements);
    hipDeviceSynchronize();

    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(5, host_array);

    // TODO 6: Free the memory
    free(host_array);
    hipFree(device_array);
    
    return 0;
}