
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX_NUM   3

#define INDEX_SUM   0
#define INDEX_MAX   1
#define INDEX_MIN   2

#define NUM_MAX         1024

#define ITEMS_NUM       (1024 * 1024)
#define BLOCK_SIZE      256

using namespace std;

// 1 thread does all compute, no atomic/sync
// thread.0 of block.0 computes everything
__global__ void kernel_no_atomics(int *data, int *results) {

    for (int i = 0; i < ITEMS_NUM; i++) {
        results[INDEX_SUM] += data[i];

        if (data[i] > results[INDEX_MAX]) {
            results[INDEX_MAX] = data[i];
        }

        if (data[i] < results[INDEX_MIN]) {
            results[INDEX_MIN] = data[i];
        }
    }
}

// ITEMS_NUM / 256 threads, ITEMS_NUM / 256 * 3 atomic calls
// thread.0 of each block does partial compute, than uses atomics to compute
__global__ void kernel_partial_atomics(int *data, int *results) {

    int start = blockIdx.x * BLOCK_SIZE;

    int localData[INDEX_NUM] = {0};
    localData[INDEX_MIN] = NUM_MAX;

    for (int i = start; i < start + BLOCK_SIZE; i++) {
        localData[INDEX_SUM] += data[i];

        if (data[i] > localData[INDEX_MAX]) {
            localData[INDEX_MAX] = data[i];
        }

        if (data[i] < localData[INDEX_MIN]) {
            localData[INDEX_MIN] = data[i];
        }
    }

    atomicAdd(&results[INDEX_SUM], localData[INDEX_SUM]);
    atomicMax(&results[INDEX_MAX], localData[INDEX_MAX]);
	atomicMin(&results[INDEX_MIN], localData[INDEX_MIN]);
}

// ITEMS_NUM threads do compute, ITEMS_NUM * 3 atomic calls
// all threads to atomics to compute
__global__ void kernel_full_atomics(int *data, int *results) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

	atomicAdd(&results[INDEX_SUM], data[idx]);
	atomicMax(&results[INDEX_MAX], data[idx]);
	atomicMin(&results[INDEX_MIN], data[idx]);
}

int main(void) {
    int expResults[INDEX_NUM];
    
    int *data = NULL;
    hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }
    
    // generate data and expected result
    expResults[INDEX_SUM] = 0;
    expResults[INDEX_MAX] = 0;
    expResults[INDEX_MIN] = NUM_MAX;
    
    for(int i = 0; i < ITEMS_NUM; i++) {
        // each generated number is lower than NUM_MAX as value
        data[i] = rand() % NUM_MAX;
        
        expResults[INDEX_SUM] += data[i];
        
        expResults[INDEX_MAX] = (data[i] > expResults[INDEX_MAX]) ?
            data[i] : expResults[INDEX_MAX];
        
        expResults[INDEX_MIN] = (data[i] < expResults[INDEX_MIN]) ?
            data[i] : expResults[INDEX_MIN];
    }
    
    int *results = NULL;
    hipMallocManaged(&results, INDEX_NUM * sizeof(int));
    if (results == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
    	return 1;
    }
   
    // compute 10 times the results
    for(int i = 0; i < 10; i++) {
        
        // init
        results[INDEX_SUM] = 0;
        results[INDEX_MAX] = 0;
        results[INDEX_MIN] = NUM_MAX;
        
#ifdef NO_ATOMIC
        kernel_no_atomics<<< 1 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef PARTIAL_ATOMIC
        kernel_partial_atomics<<< ITEMS_NUM / 256 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef FULL_ATOMIC
        kernel_full_atomics<<< ITEMS_NUM / 256 , 256 >>> (data, results);
        cudaDeviceSynchronize();
#endif
    }
    
    cout << "SUM: " << results[INDEX_SUM] << endl;
    if(results[INDEX_SUM] != expResults[INDEX_SUM]) {
        cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
    }
    
    cout << "MAX: " << results[INDEX_MAX] << endl;
    if(results[INDEX_MAX] != expResults[INDEX_MAX]) {
        cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
    }
    
    cout << "MIN: " << results[INDEX_MIN] << endl;
    if(results[INDEX_MIN] != expResults[INDEX_MIN]) {
        cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
    }
    
    hipFree(results);
    return 0;
}
