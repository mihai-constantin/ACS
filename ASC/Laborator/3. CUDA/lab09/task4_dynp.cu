
#include <hip/hip_runtime.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEM    128

using namespace std;

// workers will compute sum on first N elements
__global__ void worker(int *data, int *result)
{
    // compute sum and store in result
    for (int i = 0; i < data[threadIdx.x]; i++) {
        result[threadIdx.x] += data[i];
    }
}

// master will launch threads to compute sum on first N elements
__global__ void master(int *data, int *result, int N)
{
    // schedule worker threads
    worker<<<1, N>>>(data, result);
}

void generateData(int *data, int num) {
    srand(time(0));
    
    for(int i = 0; i < num; i++) {
        data[i] = rand() % 8 + 2;
    }
}

void print(int *data, int num) {
    for(int i = 0; i < num; i++) {
        cout << data[i] << " ";
    }
    cout << endl;
}

// TASK check
// each element result[i] should be sum of first data[i] elements of data[i]
bool checkResult(int *data, int num, int *result) {

    for(int i = 0; i < num; i++) {
        
        int sum = 0;
        for(int j = 0; j < data[i]; j++) {
            sum += data[j];
        }
        
        if(result[i] != sum) {
            cout << "Error at " << i << ", requested sum of first " 
                << data[i] << " elem, got " << result[i] << endl;
            return false;
        }
    }
    
    return true;
}

int main(int argc, char *argv[])
{
    int *data = NULL;
    hipMallocManaged(&data, NUM_ELEM * sizeof(int));

    int *result = NULL;
    hipMallocManaged(&result, NUM_ELEM * sizeof(int));
    
    generateData(data, NUM_ELEM);
    
    // schedule master threads and pass data/result/num 
    master<<<1, 1>>>(data, result, NUM_ELEM);
    hipDeviceSynchronize();
    
    print(data, NUM_ELEM);
    print(result, NUM_ELEM);
    
    if(checkResult(data, NUM_ELEM, result)) {
        cout << "Result OK" << endl;
    } else { 
        cout << "Result ERR" << endl;
    }
    
    hipFree(data);
    hipFree(result);

    return 0;
}