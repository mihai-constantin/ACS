
#include <hip/hip_runtime.h>
#include <iostream>

#define NUM_ELEM        8
#define NUM_THREADS     10

using namespace std;

__global__ void concurrentRW(int *data) {
    // NUM_THREADS try to read and write at same location
    //data[blockIdx.x] = data[blockIdx.x] + threadIdx.x;
    atomicAdd(&data[blockIdx.x], threadIdx.x);
}

int main(int argc, char *argv[]) {
    int* data = NULL;
    bool errorsDetected = false;

    hipMallocManaged(&data, NUM_ELEM * sizeof(unsigned long long int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }

    // init all elements to 0
    hipMemset(data, 0, NUM_ELEM);

    // launch kernel writes
    concurrentRW<<<NUM_ELEM, NUM_THREADS>>>(data);
    hipDeviceSynchronize();
    if (hipSuccess != hipGetLastError()) {
        return 1;
    }

    for(int i = 0; i < NUM_ELEM; i++) {
        cout << i << ". " << data[i] << endl;
        if(data[i] != (NUM_THREADS * (NUM_THREADS - 1) / 2)) {
            errorsDetected = true;
        }
    }

    if(errorsDetected) {
        cout << "Errors detected" << endl;
    } else {
        cout << "OK" << endl;
    }

    return 0;
}