#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

#define NUM_ELEM    (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
    float *host_a = 0;
    float *host_b = 0;

    // Alocare memorie unificata
    hipMallocManaged((void **) &host_a, NUM_ELEM * sizeof(float));
    hipMallocManaged((void **) &host_b, NUM_ELEM * sizeof(float));

    fill_array_random(host_b, NUM_ELEM);

    kernel_compute<<<NUM_ELEM / 256, 256>>> (host_a, host_b, NUM_ELEM);
    
    // Dealocare memorie unificata
    hipFree(host_a);
    hipFree(host_b);
    
    return 0;
}

int compute_NoUnifiedMem(void) {
    float *device_a = 0;
    float *device_b = 0;
    float *host_a = 0;
    float *host_b = 0;

    host_a = (float *) malloc(NUM_ELEM * sizeof(float));
    host_b = (float *) malloc(NUM_ELEM * sizeof(float));
    
    // Alocare memorie (GPU/VRAM)
    hipMalloc((void **) &device_a, NUM_ELEM * sizeof(float));
    hipMalloc((void **) &device_b, NUM_ELEM * sizeof(float));

    if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
        printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }

    fill_array_random(host_b, NUM_ELEM);
    
    // Copiere date host_b (CPU/RAM) => device_b (GPU/VRAM)
    hipMemcpy(device_b, host_b, NUM_ELEM * sizeof(float), hipMemcpyHostToDevice);

    kernel_compute<<<NUM_ELEM / 256, 256>>> (device_a, device_b, NUM_ELEM);
    
    // Copiere device_a (GPU/VRAM) => date host_a (CPU/RAM)
    hipMemcpy(host_a, device_a, NUM_ELEM * sizeof(float), hipMemcpyDeviceToHost);
    
    free(host_a);
    free(host_b);

    // Dealocare memorie (GPU/VRAM)
    hipFree(device_a);
    hipFree(device_b);
    
    return 0;
}

int main(void) {
#ifdef NO_UNIFIED_MEMORY
    compute_NoUnifiedMem();
#else
    compute_UnifiedMem();
#endif
}
