
#include <hip/hip_runtime.h>
#include <iostream>

#define DATA_ELEM   75
#define DB_ELEM     20
#define DB_SIZE    100

using namespace std;

struct Data {
	unsigned int d1;
	unsigned int d2;
};

union Item {
    Data data;
    unsigned long long int raw;
};

// changes only in kernel_write
// database has empty(0) and non-empty(!=0) positions, around DB_ELEM/DB_SIZE occupied
// change the kernel so each thread writes dataToAdd in an empty spot in database
// reordoring of data is permitted
// (0 1 0 1 0 ) => (t1:2, t2:3) => (2 1 1 3 0)
__global__ void kernel_write(unsigned long long int *data,
                             unsigned long long int *database,
                             int dbElemNum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long unsigned dataToAdd = data[idx];
    long long unsigned oldData;

    if(!dataToAdd) {
        return;
    }

    // logic using atomicExch to place elements in empty slots, no barriers
    while ((oldData = atomicExch(&database[idx], dataToAdd))) {
		atomicExch(&database[idx], oldData);
		idx = (idx + 1) % dbElemNum;
	}
}

// do not modify validateDB and main
//
void validateDB(unsigned long long int *database, int dbSize, int expNZElem) {

    bool isValid = true;
    int numNZElem = 0;

    for(int i = 0; i < DB_SIZE; i++) {

        Item item;
        item.raw = database[i];

        if(item.raw != 0) {
            numNZElem++;
        }

        cout << item.data.d1 << "-" << item.data.d2;
        if(item.data.d1 != item.data.d2) {
            cout << " ERR, ";
            isValid = false;
        } else {
            cout << " OK, ";
        }
    }

    if (!isValid) {
        cout << endl << "INVALID, corrupt writes" << endl;
    } else if (expNZElem != numNZElem) {
        cout << endl << "INVALID, expected database elements "
            << expNZElem << " but got " << numNZElem << endl;
    } else {
        cout << endl << "VALID" << endl;
    }
}

int main(void) {

    long long unsigned *data = 0;
    hipMallocManaged(&data, DATA_ELEM * sizeof(unsigned long long int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }

    // data to introduce into db
    for(int i = 0; i < DATA_ELEM; i++) {
        Item item;
        item.data.d1 = i + 1;
        item.data.d2 = i + 1;
        data[i] = item.raw;
    }

    long long unsigned *database = 0;
    hipMallocManaged(&database, DB_SIZE * sizeof(unsigned long long int));
    if (database == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }

    // current db is 25% full, 75% empty
    for(int i = 0; i < DB_SIZE; i++) {
        if(i % (DB_SIZE / DB_ELEM) == 0) {
            Item item;
            item.data.d1 = 1111;
            item.data.d2 = 1111;
            database[i] = item.raw;
        }
    }

    kernel_write<<<DATA_ELEM, 1>>> (data, database, DB_SIZE);
	hipDeviceSynchronize();

    validateDB(database, DB_SIZE, (DB_ELEM + DATA_ELEM));

    hipFree(data);
    hipFree(database);

    return 0;
}
