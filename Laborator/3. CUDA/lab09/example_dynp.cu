#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void childKernel() {
    printf("Hello ");
}

__global__ void parentKernel() {
    // launch child
    childKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return;
    }
    
    // wait for child to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        return;
    }
    
    printf("World!\n");
}

int main(int argc, char *argv[]) {
    // launch parent
    parentKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return 1;
    }
    
    // wait for parent to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        return 2;
    }
    
    return 0;
}
