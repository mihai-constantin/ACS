#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

__global__ void kernel_gflops(float* a, float* b, int N) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = row * N + col;

	// ~TODO~
	// Execute at least two floating point operations (e.g. +, -, *, /)
	// on the value from array a at index idx and
	// store the result in array b at index idx. 
	// NOTE: Do not change value directly in array a.

    for (int i = 0; i < 1000; ++i) {
        b[idx] = a[idx] * 2.f 
                + 1.f 
                + a[idx] * a[idx] 
                - 3.f * a[idx]
                + a[idx] / 4
                - a[idx] * 5.f;
    }
    
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);
    hipSetDevice(0);

    float *device_a = 0;
    float *device_b = 0;
    float *host_a = 0;
    float *host_b = 0;

    int N = 4096;
    int size = N * N;

    // Arrays a and b are of size N * N
    host_a = (float *) malloc(size * sizeof(float));
    host_b = (float *) malloc(size * sizeof(float));
    hipMalloc((void **) &device_a, size * sizeof(float));
    hipMalloc((void **) &device_b, size * sizeof(float));

    if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
        printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }

    // Populate array a randomly
    fill_array_random(host_a, size);
    hipMemcpy(device_a, host_a, size * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    // Create two cuda events (start and stop)
    // by using the hipEventCreate function.
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel_gflops<<<size / 256, 256>>> (device_a, device_b, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    float seconds = ms / pow((float) 10, 3);
    printf("seconds: %.3f\n", seconds);

    // Set num_ops to the number of floating point operations
    // done in the kernel multiplied with the size of the matrix.
    long num_ops = 10000 * (long)size;

    float gflops = (float) num_ops / seconds / 1e+9;
    printf("GFLOPS: %.3f\n", gflops);

    free(host_a);
    free(host_b);
    hipFree(device_a);
    hipFree(device_b);
    
    return 0;
}
