#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    const int num_bytes = BUF_32M * sizeof(int);

    // TODO 1: Allocate the host's arrays with the specified number of elements:
    // host_array_a => 32M
    // host_array_b => 32M

    host_array_a = (int *) malloc(num_bytes);
    host_array_b = (int *) malloc(num_bytes);

    // TODO 2: Allocate the device's arrays with the specified number of elements:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M
    hipMalloc((void **) &device_array_a, num_bytes);
    hipMalloc((void **) &device_array_b, num_bytes);
    hipMalloc((void **) &device_array_c, BUF_2M * sizeof(int));

    // Check for allocation errors
    if (!host_array_a || !host_array_b || 
        !device_array_a || !device_array_b || !device_array_c) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device
    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: hipMemcpy

    int offset, i;
    for(i = 0; i < BUF_32M / BUF_2M; i++) {
        offset = i * BUF_2M;
        hipMemcpy(device_array_c + offset, device_array_a + offset, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(device_array_a + offset, device_array_b + offset, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(device_array_b + offset, device_array_c + offset, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
    }

    // TODO 5: Copy from device to host
    hipMemcpy(host_array_a, device_array_a, num_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(host_array_b, device_array_b, num_bytes, hipMemcpyDeviceToHost);

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory
    free(host_array_a);
    free(host_array_b);
    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);

    return 0;

}

