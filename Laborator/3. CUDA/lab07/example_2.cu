
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

/**
 * This kernel computes the function f(x) = 2x + 1/(x + 1) for each
 * element in the given array.
 */
__global__ void vector_add(const float *a, float *b, const size_t n) {
  	// Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  	// Avoid accessing out of bounds elements
  	if (i < n) {
    	b[i] = 2.0 * a[i] + 1.0 / (a[i] + 1.0);
  	}
}


int main(void) 
{  
  // Declare variable to represent ~1M float values and
  // computes the amount of bytes necessary to store them
  const int num_elements = 1 << 16;
  const int num_bytes = num_elements * sizeof(float);

	// Declaring the 'host arrays': a host array is the classical
	// array (static or dynamically allocated) we worked before.
  float *host_array_a = 0;
  float *host_array_b = 0;
	
	// Declaring the 'device array': this array is the equivalent
	// of classical array from C, but specially designed for the GPU
	// devices; we declare it in the same manner, but the allocation
	// process is going to be different
  float *device_array_a = 0;
  float *device_array_b = 0;

  // Allocating the host array
  host_array_a = (float *) malloc(num_bytes);
	host_array_b = (float *) malloc(num_bytes);

	// Allocating the device's array; notice that we use a special
	// function named cudaMalloc that takes the reference of the
	// pointer declared above and the number of bytes.
  hipMalloc((void **) &device_array_a, num_bytes);
	hipMalloc((void **) &device_array_b, num_bytes);

  // If any memory allocation failed, report an error message
  if (host_array_a == 0 || host_array_b == 0|| device_array_a == 0 || device_array_b == 0) {
    return 1;
  }

	// Initialize the host array by populating it with float values  
	for (int i = 0; i < num_elements; ++i) {
  	host_array_a[i] = (float) i;
	}

	// Copying the host array to the device memory space; notice the
	// parameters of the cudaMemcpy function; the function default
	// signature is cudaMemcpy(dest, src, bytes, flag) where
	// the flag specifies the transfer type.
	//
	// host -> device: cudaMemcpyHostToDevice
	// device -> host: cudaMemcpyDeviceToHost
	// device -> device: cudaMemcpyDeviceToDevice
	hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);

	// Compute the parameters necessary to run the kernel: the number
	// of blocks and the number of threads per block; also, deal with
	// a possible partial final block
  const size_t block_size = 256;
  size_t blocks_no = num_elements / block_size;

  if (num_elements % block_size) 
    ++blocks_no;

  // Launch the kernel
  vector_add<<<blocks_no, block_size>>>(device_array_a, device_array_b, num_elements);

  // Copy the result back to the host memory space
  hipMemcpy(host_array_b, device_array_b, num_bytes, hipMemcpyDeviceToHost);

	// Print out the first 10 results
  for (int i = 0; i < 10; ++i) {
    printf("Result %d: 2 * %1.1f + 1.0/(%1.1f + 1.0)= %1.3f\n", 
    i, host_array_a[i], host_array_a[i], host_array_b[i]);
	}

  // Deallocate memory
  free(host_array_a);
  free(host_array_b);
  hipFree(device_array_a);
  hipFree(device_array_b);

  return 0;
}

