
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_example(int value) {
	/**
	 * This is a kernel; a kernel is a piece of code that
	 * will be executed by each thread from each block in
	 * the GPU device.
	 */
	printf("[GPU] Hello from the GPU!\n");
	printf("[GPU] The value is %d\n", value);
}

int main(void) {
	/**
	 * Here, we declare and/or initialize different values or we
	 * can call different functions (as in every C/C++ program);
	 * In our case, here we also initialize the buffers, copy
	 * local data to the device buffers, etc (you'll see more about
	 * this in the following exercises).
	 */
	int nDevices;
	printf("[HOST] Hello from the host!\n");

	/**
	 * Get the number of compute-capable devices. See more info 
	 * about this function in the Cuda Toolkit Documentation.
	 */
	hipGetDeviceCount(&nDevices);
	printf("[HOST] You have %d CUDA-capable GPU(s)\n", nDevices);

	/** 
	 * Launching the above kernel with a single block, each block
	 * with a single thread. The syncrhonize and the checking functions
	 * assures that everything works as expected.
	 */
	kernel_example<<<1,1>>>(25);
	hipDeviceSynchronize();

	/**
	 * Here we can also deallocate the allocated memory for the device
	 */
  	return 0;
}

